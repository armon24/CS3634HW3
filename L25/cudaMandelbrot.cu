#include "hip/hip_runtime.h"
/* 

To compile:

   gcc -O3 -o mandelbrot mandelbrot.c -lm

To create an image with 4096 x 4096 pixels

    ./mandelbrot 4096 4096 

*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI);

// Q2a: add include for CUDA header file here:

#define MXITER 1000

typedef struct {
  
  double r;
  double i;
  
}complex_t;

// return iterations before z leaves mandelbrot set for given c
int testpoint(complex_t c){

  int iter;
  complex_t z;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){  
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return iter;
    }
  }
  return iter; 
}

// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void mandelbrot(int Nre, int Nim, complex_t cmin, complex_t dc, float *count){ 

  int xthreadIndex = threadIdx.x;
  int xblockIndex  = blockIdx.x;
  int xthreadCount = blockDim.x;

  int ythreadIndex = threadIdx.y;
  int yblockIndex  = blockIdx.y;
  int ythreadCount = blockDim.y;

  int nX = xthreadIndex + xthreadCount*xblockIndex;
  int nY = ythreadIndex + ythreadCount*yblockIndex;

  complex_t c;

  c.r = cmin.r + dc.r*nX;
  c.i = cmin.i + dc.i*nY;
      
  count[nX+nY*Nre] = (float) testpoint(c);

}

int main(int argc, char **argv){

  // to create a 4096x4096 pixel image 
  // usage: ./mandelbrot 4096 4096 

  int Nre = (argc==3) ? atoi(argv[1]): 4096;
  int Nim = (argc==3) ? atoi(argv[2]): 4096;

  int N = Nre*Nim;
  // Q2b: set the number of threads per block and the number of blocks here:

  //HOST
  float *h_a = (float*) malloc(N*sizeof(float));

  //DEVICE
  float *c_a;

  //CUDAMALLOC
  hipMalloc(&c_a, N*sizeof(float));

  //CUDAMEMCPY
  hipMemcpy(c_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);

  dim3 TPB(16,16,1); // Bx * By threads in thread-block
  dim3 BPG((Nre+15)/16,(Nim+15)/16,1); // Gx * Gy grid of thread-blocks


  // storage for the iteration counts
  float *count;
  count = (float*) malloc(Nre*Nim*sizeof(float));

  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;

  complex_t cmin; 
  complex_t cmax;
  complex_t dc;

  cmin.r = centRe - 0.5*diam;
  cmax.r = centRe + 0.5*diam;
  cmin.i = centIm - 0.5*diam;
  cmax.i = centIm + 0.5*diam;

  //set step sizes
  dc.r = (cmax.r-cmin.r)/(Nre-1);
  dc.i = (cmax.i-cmin.i)/(Nim-1);

  clock_t start = clock(); //start time in CPU cycles

  // compute mandelbrot set
  mandelbrot <<< BPG, TPB >>> (Nre, Nim, cmin, dc, count); 

  hipMemcpy(h_a, c_a, N*sizeof(float), hipMemcpyDeviceToHost);
  
  // copy from the GPU back to the host here

  clock_t end = clock(); //start time in CPU cycles
  
  // print elapsed time
  printf("elapsed = %f\n", ((double)(end-start))/CLOCKS_PER_SEC);

  // output mandelbrot to ppm format image
  printf("Printing mandelbrot.ppm...");
  writeMandelbrot("mandelbrot.ppm", Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  


/* Output data as PPM file */
void saveppm(const char *filename, unsigned char *img, int width, int height){

  /* FILE pointer */
  FILE *f;
  
  /* Open file for writing */
  f = fopen(filename, "wb");
  
  /* PPM header info, including the size of the image */
  fprintf(f, "P6 %d %d %d\n", width, height, 255);

  /* Write the image data to the file - remember 3 byte per pixel */
  fwrite(img, 3, width*height, f);

  /* Make sure you close the file */
  fclose(f);
}



int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI){

  int n, m;

  unsigned char *rgb   = (unsigned char*) calloc(3*width*height, sizeof(unsigned char));
  
  for(n=0;n<height;++n){
    for(m=0;m<width;++m){
      int id = m+n*width;

      int I = (int) (768*sqrt((double)(img[id]-minI)/(maxI-minI)));
      
      // change this to change palette
      if(I<256)      rgb[3*id+2] = 255-I;
      else if(I<512) rgb[3*id+1] = 511-I;
      else if(I<768) rgb[3*id+0] = 767-I;
      else if(I<1024) rgb[3*id+0] = 1023-I;
      else if(I<1536) rgb[3*id+1] = 1535-I;
      else if(I<2048) rgb[3*id+2] = 2047-I;

    }
  }

  saveppm(fileName, rgb, width, height);

  free(rgb);
}


